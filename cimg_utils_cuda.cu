#include "hip/hip_runtime.h"
/*
*   cimg.cpp
*   part of image blur software using CUDA
*   for CSC 630 with Dr. Zhang
*   Created by Dan McGonigle, 11/21/2019
*
*   This program implements CImg functionality for the image blur software
*/

#define cimg_OS 1
#define cimg_display 0
#include "CImg.h" 
#include "cimg_utils.h"
#include <iostream> 
#include <stdlib.h>
#include <vector>

namespace cl=cimg_library;
 
/*
*           CUDA FUNCTIONS
*/

/*
*   Split the channels apart.
*   uchar4 is a built-in vector struct with special allignment:
*   https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#vector-types
*
*   c_red, c_green and c_blue are the color channels to be returned from this kernel.
*/
/*  Kept getting segmentation faults from merge_channels
__global__
void split_channels(const uchar4* const image, int rows, int cols, 
                    unsigned char* const c_red,
                    unsigned char* const c_green, 
                    unsigned char* const c_blue)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    //  Set bounds for channel
    if (col >= cols || row >= rows) 
    {
        return;
    }
    int index = row * cols + col;
    c_red[index] = image[index].x;
    c_green[index] = image[index].y;
    c_blue[index] = image[index].z;
}
*/

/*
*   Blur
*/
__global__
void apply_blur_cuda(const unsigned char* const input, unsigned char* const output,
                   int rows, int cols, const float* const filter, const int filterSize)
{
    
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    //  Set bounds for blur filter (no padding)
    if (col >= cols - filterSize || row >= rows - filterSize || col < filterSize || row < filterSize) 
    {
        return;
    }
    int index = row * cols + col;
    /*
    float sum = 0.0;

    for (int frow = row - filterSize; frow <= row + filterSize; frow++)
    {
        for (int fcol = col - filterSize; fcol <= col + filterSize; fcol++)
        {
            //  vector row and column index
            int vrow = frow - row + filterSize;
            int vcol = fcol - col + filterSize;
            sum += filter[vrow*filterSize+vcol] * input[frow*cols+fcol];
        }
    }
    output[index] = sum;
    */

    output[index] = input[index] / 2;
}

/*
*   Bring channels back together
*/
/*  Kept getting segmentation faults on merge_channels
__global__
void merge_channels(const unsigned char* const c_red, 
                    const unsigned char* const c_green, 
                    const unsigned char* const c_blue,
                    uchar4* const output, int rows, int cols)
{
    const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int thread_1D_pos = thread_2D_pos.y * cols + thread_2D_pos.x;
    //  Set bounds for channel dims
    if (thread_2D_pos.x >= cols || thread_2D_pos.y >= rows)
    {
        return;
    }

    unsigned char red = c_red[thread_1D_pos];
    unsigned char green = c_green[thread_1D_pos];
    unsigned char blue = c_blue[thread_1D_pos];

    //  Alpha 255 => no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);

    output[thread_1D_pos] = outputPixel;
}
*/

/*
*       END CUDA KERNELS ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
*/

//  Cuda blur
cl::CImg<unsigned char> blur_cuda( cl::CImg<unsigned char> image , int filterSize )
{
    //  Create filter 2D array
    float **filter = new float*[2*filterSize + 1];
    getFilter(filter, filterSize);
    printFilter(filter, filterSize);

    //  Set block size (number of threads per block), then grid size (number of blocks per kernel)
    const dim3 block_size(16,16,1);
    const dim3 grid_size(image.width()/block_size.x+1, image.height()/block_size.y+1,1);

    //  Variables to hold information on cuda memory
    unsigned char *cuda_red, *cuda_green, *cuda_blue;
    float *cuda_filter;
    /*  A NOTE ABOUT *ptr = CImg.data():
    'T *ptr = img.data()' gives you the pointer to the first value of the image 'img'. 
    The overall size of the used memory for one instance image (in bytes) is then 'width*height*depth*dim*sizeof(T)'.
    Now, the ordering of the pixel values in this buffer follows these rules : 
        The values are not interleaved, and are ordered first along the X,Y,Z and V axis respectively 
        (corresponding to the width,height,depth,dim dimensions), 
        starting from the upper-left pixel to the bottom-right pixel of the instane image, with a classical scanline run.
        So, a color image with dim=3 and depth=1, will be stored in memory as :R1R2R3R4R5R6......G1G2G3G4G5G6.......B1B2B3B4B5B6.... 
        (i.e following a 'planar' structure)and not as R1G1B1R2G2B2R3G3B3... (interleaved channels).
    */
    //unsigned char *cuda_image = image.data();
    //unsigned char *cuda_image;

    //  flatten filter
    //cuda_filter = &(filter[0][0]);

    //  Declare GPU memory pointers
    unsigned char *cuda_red_blurred, *cuda_green_blurred, *cuda_blue_blurred;//, *cuda_image_blurred;

    //  Allocate memory to cuda
    int channel_size = image.get_channel(0).size();
    std::cout << "Channel size: " << channel_size << std::endl;
    std::cout << "Unsigned char size: " << sizeof(unsigned char) << std::endl;
    std::cout << "Int size: " << sizeof(int) << std::endl;

    hipMalloc((void**)&cuda_red, sizeof(unsigned char) * channel_size);
    hipMalloc((void**)&cuda_red_blurred, sizeof(unsigned char) * channel_size);
    hipMalloc((void**)&cuda_green, sizeof(unsigned char) * channel_size);
    hipMalloc((void**)&cuda_green_blurred, sizeof(unsigned char) * channel_size);
    hipMalloc((void**)&cuda_blue, sizeof(unsigned char) * channel_size);
    hipMalloc((void**)&cuda_blue_blurred, sizeof(unsigned char) * channel_size);
    hipMalloc((void**)&cuda_filter, sizeof(float) * (2*filterSize+1) * (2*filterSize+1));
    //hipMalloc((void**)&cuda_image, sizeof(uchar4) * image.size());
    //hipMalloc((void**)&cuda_image_blurred, sizeof(uchar4) * image.size());
    //hipMalloc((void**)&cuda_image, sizeof(unsigned char) * image.size());
    //hipMalloc((void**)&cuda_image_blurred, sizeof(unsigned char) * image.size());

    //  Transfer image and filter to GPU
    //hipMemcpy(cuda_image, image.data(), sizeof(uchar4) * image.size(), hipMemcpyHostToDevice);
    //hipMemcpy(cuda_image, image.data(), sizeof(unsigned char) * image.size(), hipMemcpyHostToDevice);
    hipMemcpy(cuda_filter, &(filter[0][0]), sizeof(float) * (2*filterSize+1) * (2*filterSize+1), hipMemcpyHostToDevice);
    //hipMemcpy(cuda_red, cuda_red, sizeof(unsigned char) * image.height() * image.width(), hipMemcpyHostToDevice);
    //hipMemcpy(cuda_green, cuda_green, sizeof(unsigned char) * image.height() * image.width(), hipMemcpyHostToDevice);
    //hipMemcpy(cuda_blue, cuda_blue, sizeof(unsigned char) * image.height() * image.width(), hipMemcpyHostToDevice);

/*  Kept getting segmentation faults on merge_channels
    //  Split channels
    split_channels<<<grid_size, block_size>>> (reinterpret_cast<uchar4*>(cuda_image),
                                                image.height(), 
                                                image.width(), 
                                                cuda_red, 
                                                cuda_green, 
                                                cuda_blue);
*/

    hipMemcpy(cuda_red, (unsigned char*)image.get_channel(0), sizeof(unsigned char) * channel_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_green, (unsigned char*)image.get_channel(1), sizeof(unsigned char) * channel_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_blue, (unsigned char*)image.get_channel(2), sizeof(unsigned char) * channel_size, hipMemcpyHostToDevice);
    
    //  Apply blur
    apply_blur_cuda<<<grid_size, block_size>>> (cuda_red, 
                                                cuda_red_blurred, 
                                                image.height(), 
                                                image.width(), 
                                                cuda_filter, 
                                                filterSize);
    apply_blur_cuda<<<grid_size, block_size>>> (cuda_green, 
                                                cuda_green_blurred, 
                                                image.height(), 
                                                image.width(), 
                                                cuda_filter, 
                                                filterSize);
    apply_blur_cuda<<<grid_size, block_size>>> (cuda_blue, 
                                                cuda_blue_blurred, 
                                                image.height(), 
                                                image.width(), 
                                                cuda_filter, 
                                                filterSize);

    unsigned char *red_blurred =  (unsigned char*)malloc (sizeof(unsigned char) * channel_size);
    unsigned char *green_blurred = (unsigned char*)malloc (sizeof(unsigned char) * channel_size);
    unsigned char *blue_blurred = (unsigned char*)malloc (sizeof(unsigned char) * channel_size);
    hipMemcpy(&red_blurred, cuda_red_blurred, sizeof(unsigned char) * channel_size, hipMemcpyDeviceToHost);
    hipMemcpy(&green_blurred, cuda_green_blurred, sizeof(unsigned char) * channel_size, hipMemcpyDeviceToHost);
    hipMemcpy(&blue_blurred, cuda_blue_blurred, sizeof(unsigned char) * channel_size, hipMemcpyDeviceToHost);
    //hipMemcpy(&image, cuda_image, image.size(), hipMemcpyDeviceToHost);


    //  loop through image to get blurred pixels
    for (int row = 0; row < image.height(); row++)
    {
        for (int col = 0; col < image.width(); col++)
        {
            image(col, row, 0, 0) = red_blurred[row*image.width()+col];
            image(col, row, 0, 1) = green_blurred[row*image.width()+col];
            image(col, row, 0, 2) = blue_blurred[row*image.width()+col];
        }
    }


/*  Kept getting segmentation faults on merge_channels
    //  Merge channels
    merge_channels<<<grid_size, block_size>>> (cuda_red_blurred, 
                                                cuda_green_blurred, 
                                                cuda_blue_blurred,
                                                reinterpret_cast<uchar4*>(cuda_image_blurred),
                                                image.height(),
                                                image.width());
*/

    //hipMemcpy(&image, cuda_image, image.size(), hipMemcpyDeviceToHost);

    char char_break;
    std::cout << "Pause for character break before freeing up memory: ";
    std::cin >> char_break;

    //  Free up space
    hipFree(cuda_red);
    hipFree(cuda_red_blurred);
    hipFree(cuda_green);
    hipFree(cuda_green_blurred);
    hipFree(cuda_blue);
    hipFree(cuda_blue_blurred);
    //hipFree(cuda_image);
    //hipFree(cuda_image_blurred);
    hipFree(cuda_filter);

    std::cout << "Pause for character break after freeing up memory: ";
    std::cin >> char_break;

    return image;
}


